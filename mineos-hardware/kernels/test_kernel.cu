
#include <hip/hip_runtime.h>
// Test CUDA kernel for verifying GPU framework
extern "C" {

__global__ void vector_add(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void fill_buffer(float* buffer, float value, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        buffer[idx] = value;
    }
}

__global__ void multiply_add(const float* a, const float* b, float scalar, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] * scalar + b[idx];
    }
}

// Simple hash function for testing (not cryptographic!)
__global__ void simple_hash(const uint32_t* input, uint32_t* output, uint32_t nonce_start, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        uint32_t nonce = nonce_start + idx;
        uint32_t hash = input[0] ^ nonce;
        
        // Simple mixing
        hash = hash * 0x85ebca6b;
        hash = hash ^ (hash >> 13);
        hash = hash * 0xc2b2ae35;
        hash = hash ^ (hash >> 16);
        
        output[idx] = hash;
    }
}

}